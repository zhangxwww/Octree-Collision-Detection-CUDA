#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <vector>
#include <iostream>

#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>


#include "Shader.h"
#include "KeyboardController.h"

#include "Constant.h"



KeyboardController keyboardController;

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode);
GLFWwindow* init();
void initSphere();

glm::vec3 positions[] = {
    glm::vec3(-1.0f, 0.0f, -1.0f),
    glm::vec3(1.0f, 0.0f, 1.0f),
    glm::vec3(-1.0f, 0.0f, 0.0f)
};

std::vector<float> sphereVertices;
std::vector<int> sphereIndices;

int main() {
    GLFWwindow* window = init();
    if (window == nullptr) {
        return 0;
    }

    initSphere();

    unsigned int VBO, VAO;
    glGenVertexArrays(1, &VAO);
    glGenBuffers(1, &VBO);
    glBindVertexArray(VAO);
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sphereVertices.size() * sizeof(float), &sphereVertices[0], GL_STATIC_DRAW);

    GLuint element_buffer_object;//EBO
    glGenBuffers(1, &element_buffer_object);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, element_buffer_object);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sphereIndices.size() * sizeof(int), &sphereIndices[0], GL_STATIC_DRAW);

    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), (void*)0);
    glEnableVertexAttribArray(0);

    glBindBuffer(GL_ARRAY_BUFFER, 0);
    glBindVertexArray(0);

    Shader shader;
    shader.loadFromFile("vertex.glsl", "fragment.glsl");

    float lastTime = glfwGetTime();
    while (!glfwWindowShouldClose(window) && !keyboardController.shouldClose()) {
        float currentTime = glfwGetTime();
        float deltaTime = currentTime - lastTime;
        lastTime = currentTime;

        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        shader.use();

        keyboardController.step(deltaTime);

        glm::mat4 projection = glm::perspective(glm::radians(45.0f), (float)(WIDTH) / HEIGHT, 1.0f, 100.0f);
        glm::mat4 view = glm::lookAt(keyboardController.getEyePos(), glm::vec3(0.0f, 0.0f, 0.0f), glm::vec3(0.0f, 1.0f, 0.0f));

        for (int i = 0; i < 3; ++i) {
            glm::mat4 model = glm::mat4(1.0f);
            model = glm::translate(model, positions[i]);
            model = glm::scale(model, glm::vec3(RADIUS, RADIUS, RADIUS));

            shader.setMat4("m", model);
            shader.setMat4("v", view);
            shader.setMat4("p", projection);
            shader.setVec3("color", glm::vec3(1.0f, 1.0f, 1.0f));

            glEnable(GL_CULL_FACE);
            glCullFace(GL_BACK);
            glBindVertexArray(VAO);
            glPointSize(2);
            glDrawElements(GL_POINTS, X_SEGMENTS * Y_SEGMENTS * 6, GL_UNSIGNED_INT, 0);
        }
       
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    glDeleteVertexArrays(1, &VAO);
    glDeleteBuffers(1, &VBO);
    glDeleteBuffers(1, &element_buffer_object);

    glfwTerminate();
    return 0;
}

void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode) {
    keyboardController.callback(window, key, scancode, action, mode);
}

GLFWwindow* init() {
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
    glfwWindowHint(GLFW_RESIZABLE, false);

    auto window = glfwCreateWindow(WIDTH, HEIGHT, "Collision Detection", nullptr, nullptr);
    if (window == nullptr)
    {
        std::cout << "Failed to Create OpenGL Context" << std::endl;
        glfwTerminate();
        return nullptr;
    }
    glfwMakeContextCurrent(window);
    glfwSetKeyCallback(window, key_callback);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return nullptr;
    }
    glViewport(0, 0, WIDTH, HEIGHT);
    return window;
}

void initSphere() {

    for (int y = 0; y <= Y_SEGMENTS; y++)
    {
        for (int x = 0; x <= X_SEGMENTS; x++)
        {
            float xSegment = (float)x / (float)X_SEGMENTS;
            float ySegment = (float)y / (float)Y_SEGMENTS;
            float xPos = std::cos(xSegment * 2.0f * PI) * std::sin(ySegment * PI);
            float yPos = std::cos(ySegment * PI);
            float zPos = std::sin(xSegment * 2.0f * PI) * std::sin(ySegment * PI);
            sphereVertices.push_back(xPos);
            sphereVertices.push_back(yPos);
            sphereVertices.push_back(zPos);
        }
    }

    for (int i = 0; i < Y_SEGMENTS; i++)
    {
        for (int j = 0; j < X_SEGMENTS; j++)
        {
            sphereIndices.push_back(i * (X_SEGMENTS + 1) + j);
            sphereIndices.push_back((i + 1) * (X_SEGMENTS + 1) + j);
            sphereIndices.push_back((i + 1) * (X_SEGMENTS + 1) + j + 1);
            sphereIndices.push_back(i * (X_SEGMENTS + 1) + j);
            sphereIndices.push_back((i + 1) * (X_SEGMENTS + 1) + j + 1);
            sphereIndices.push_back(i * (X_SEGMENTS + 1) + j + 1);
        }
    }
}
